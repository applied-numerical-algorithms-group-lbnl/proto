#include "../include/Proto.H"
#include "hip/hip_runtime.h"
#include <chrono>

using namespace Proto;

namespace {

    #ifdef PROTO_CUDA
    #define CHECK_DEVICE_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUA error in %s:%d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)
    #else
    #define CHECK_DEVICE_ERROR(call)
    #endif

    DisjointBoxLayout testLayout(int domainSize, Point boxSize, std::set<Point> skipPatches)
    {
        Box domainBox = Box::Cube(domainSize); 
        Box patchBox = domainBox.coarsen(boxSize);
        std::vector<Point> patches;
        for (auto patch : patchBox)
        {
            bool skip = skipPatches.find(patch) != skipPatches.end();
            if (!skip) { patches.push_back(patch); }
        }
        std::array<bool, DIM> periodicity;
        periodicity.fill(true);
        ProblemDomain domain(domainBox, periodicity);
        return DisjointBoxLayout(domain, patches, boxSize);
    }
}

double profileCPU(const DisjointBoxLayout& layout)
{
    using namespace chrono;

    LevelBoxData<double, 1, HOST> data(layout, Point::Zeros());
    for (auto iter : data)
    {
        data[iter].setVal(iter.local());
    }

    auto start = high_resolution_clock::now();
    auto absMaxValue = data.absMax();
    auto end = high_resolution_clock::now();
    auto elapsed = duration_cast<milliseconds>(end - start);

    return elapsed.count();
}

#ifdef PROTO_CUDA
double profileCuda(const DisjointBoxLayout& layout)
{
    LevelBoxData<double, 1, DEVICE> data(layout, Point::Zeros());
    for (auto iter : data)
    {
        data[iter].setVal(iter.local());
    }
    hipEvent_t start, stop;

    CHECK_DEVICE_ERROR(hipEventCreate(&start));
    CHECK_DEVICE_ERROR(hipEventCreate(&stop));

    CHECK_DEVICE_ERROR(hipEventRecord(start));

    auto absMaxValue = data.absMax();
    CHECK_DEVICE_ERROR(hipGetLastError());

    CHECK_DEVICE_ERROR(hipEventRecord(stop));
    CHECK_DEVICE_ERROR(hipEventSynchronize(stop));

    double milliseconds = 0;
    CHECK_DEVICE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));

    return milliseconds;
}
#endif

int main(int argc, char** argv)
{
    int domainSize = 512;
    auto boxSize = 32*Point::Ones();
    std::set<Point> skipPatches;
    auto layout = testLayout(domainSize, boxSize, skipPatches);

    auto cpuTime = profileCPU(layout);
    std::cout << "CPU Time: " << std::endl;
    #ifdef PROTO_CUDA
    auto cudaTime = profileCuda(layout);
    std::cout << "Cuda Time: " << std::endl;
    #endif

}
