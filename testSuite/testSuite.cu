#include <iostream>
#include <test_fusion_bc.cu>
#include <test_forall.cu>
#include <test_stack.cu>
#include <test_reduction.cu>


template<typename Func>
void do_test(std::string a_str, Func &fun)
{
  std::cout << "Do " << a_str << std::endl;
  bool b = fun();
  if(b) std::cout << "-> passed " << a_str << std::endl;
  else std::cout << "-> failed " << a_str << std::endl;
}

int main()
{
  hipSetDevice(1);
  do_test("test_fusion_bc",   run_test_fusion_bc); 
  do_test("test_forall",      run_test_forall); 
  do_test("test_forall_i",    run_test_forall_p); 
  do_test("test_forall_p",    run_test_forall_i); 
  do_test("test_stack_using", run_test_stack_using); 
  do_test("test_stack_free",  run_test_stack_free); 
  do_test("test_stack_empty",  run_test_stack_empty); 
  do_test("test_stack_reset",  run_test_stack_empty); 
  do_test("test_reduction_min_linear_init_1",  test_reduction_min_linear_init_1); 
  do_test("test_reduction_min_linear_init_minus_2",  test_reduction_min_linear_init_minus_2); 
  do_test("test_reduction_max_linear_init_1",  test_reduction_max_linear_init_1); 
  do_test("test_reduction_max_linear_init_minus_2",  test_reduction_max_linear_init_minus_2); 
  return 0;  
}
