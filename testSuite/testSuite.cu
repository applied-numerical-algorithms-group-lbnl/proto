#include <iostream>
#include <test_fusion_bc.cu>
#include <test_forall.cu>
#include <test_stack.cu>


template<typename Func>
void do_test(std::string a_str, Func &fun)
{
  std::cout << "Do " << a_str << std::endl;
  bool b = fun();
  if(b) std::cout << "-> passed " << a_str << std::endl;
  else std::cout << "-> failed " << a_str << std::endl;
}

int main()
{
  hipSetDevice(1);
  do_test("test_fusion_bc",   run_test_fusion_bc); 
  do_test("test_forall",      run_test_forall); 
  do_test("test_forall_i",    run_test_forall_p); 
  do_test("test_forall_p",    run_test_forall_i); 
  do_test("test_stack_using", run_test_stack_using); 
  do_test("test_stack_free",  run_test_stack_free); 
  return 0;  
}
