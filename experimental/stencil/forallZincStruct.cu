#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <cmath>

#include <vector>
#include <memory>

#include <iostream>
#include <fstream>
#include <sstream>


#include "Proto.H"
using std::cout;
using std::endl;
using std::vector;
using std::shared_ptr;
using namespace Proto;
constexpr unsigned int NUMCOMPS=DIM+2;
typedef Var<double,NUMCOMPS> State;

//arbitrary int for number of multiplies to replace square root in riemann

/**/
void
parseCommandLine(unsigned int& a_nmult, unsigned int & a_nx, unsigned int& a_maxgrid, unsigned int & a_numapplies, int argc, char* argv[])
{
  //defaults
  a_nx = 256;
  a_numapplies = 100;
  a_nmult = 100;
  a_maxgrid = 64;
  cout << "kernel timings of various foralls ([] shows defaults)" << endl;
  cout << "usage:  " << argv[0] << " -n nx[256] -m max_grid[128] -a num_iterations[100] -s max_streams[32] -t num_multiplies[100]" << endl;
  for(int iarg = 0; iarg < argc-1; iarg++)
  {
    if(strcmp(argv[iarg],"-n") == 0)
    {
      a_nx = atoi(argv[iarg+1]);
    }
    else if(strcmp(argv[iarg], "-a") == 0)
    {
      a_numapplies = atoi(argv[iarg+1]);
    }
    else if(strcmp(argv[iarg], "-m") == 0)
    {
      a_maxgrid = atoi(argv[iarg+1]);
    }
    else if(strcmp(argv[iarg], "-t") == 0)
    {
      a_nmult = atoi(argv[iarg+1]);
    }
  }
  cout << "nx          = " << a_nx << endl;
  cout << "num_applies = " << a_numapplies << endl; 
  cout << "maxgrid     = " << a_maxgrid << endl;
  cout << "num_mult    = " << a_nmult << endl;
}


PROTO_KERNEL_START
void upwindStateF(State& a_out,
                  const State& a_low,
                  const State& a_high,
                  int   a_dir,
                  double a_gamma,
                  unsigned int a_nmult)
{
  const double& rhol = a_low(0);
  const double& rhor = a_high(0);
  const double& ul = a_low(a_dir+1);
  const double& ur = a_high(a_dir+1);
  const double& pl = a_low(NUMCOMPS-1);
  const double& pr = a_high(NUMCOMPS-1);
  double gamma = a_gamma;
  //2
  double rhobar = (rhol + rhor)*.5;
  //2
  double pbar = (pl + pr)*.5;
  //2
  double ubar = (ul + ur)*.5;
  //took this one out for a bunch of multiplies so
  //I can have flops I can count
//  double cbar = sqrt(gamma*pbar/rhobar);
  //2
  double cbar = gamma*pbar/rhobar;
  //NMULT
  for(int iter = 0; iter < a_nmult; iter++)
  {
    cbar *= pbar;
  }
  //7
  double pstar = (pl + pr)*.5 + rhobar*cbar*(ul - ur)*.5;
  //7
  double ustar = (ul + ur)*.5 + (pl - pr)/(2*rhobar*cbar);
  int sign;
  if (ustar > 0) 
  {
    sign = -1;
    for (int icomp = 0;icomp < NUMCOMPS;icomp++)
    {
      a_out(icomp) = a_low(icomp);
    }
  }
  else
  {
    sign = 1;
    for (int icomp = 0;icomp < NUMCOMPS;icomp++)
    {
      a_out(icomp) = a_high(icomp);
    }
  }
  //2
  if (cbar + sign*ubar > 0)
  {
    //4
    a_out(0) += (pstar - a_out(NUMCOMPS-1))/(cbar*cbar);
    a_out(a_dir+1) = ustar;
    a_out(NUMCOMPS-1) = pstar;
  }
  //I get 28 + NMULT
}
PROTO_KERNEL_END(upwindStateF, upwindState)


PROTO_KERNEL_START
void doNothingF(State& a_out,
                const State& a_low,
                const State& a_high,
                int   a_dir,
                double a_gamma,
                unsigned int a_nmult)
{
}
PROTO_KERNEL_END(doNothingF, doNothing)


PROTO_KERNEL_START
void doNothingOneBDF(State& a_out)
{
}
PROTO_KERNEL_END(doNothingOneBDF, doNothingOneBD)

struct DoNothingStruct 
{ 
  __device__ void op(State& a_out,
                     const State& a_low,
                     const State& a_high,
                     int   a_dir,
                     double a_gamma,
                     unsigned int a_nmult)
  { 
    return doNothing(a_out, a_low, a_high, a_dir, a_gamma, a_nmult);
  }
};


struct UpwindStruct 
{ 
  __device__ void op(State& a_out,
                     const State& a_low,
                     const State& a_high,
                     int   a_dir,
                     double a_gamma,
                     unsigned int a_nmult)
  { 
    return upwindState(a_out, a_low, a_high, a_dir, a_gamma, a_nmult);
  }
};
///proxies for Chombo-style SPMD functions
unsigned int CH_numProc()
{
  return 1;
}

unsigned int CH_procID()
{
  return 0;
}

///boxes that cover a domain box
class DisjointBoxLayout
{
private:
  struct localData
  {
    Box                   m_coarsenedDom;
    vector<unsigned int>  m_procs;
    vector<unsigned int>  m_localBoxes;
    unsigned int          m_maxgrid;
  };

  //this is to make this a ref-counted object
  shared_ptr<localData> m_internals;


public:

  DisjointBoxLayout()
  {;}

  ///
  DisjointBoxLayout(const Box& a_domain, const unsigned int& a_maxgrid)
  {
    define(a_domain, a_maxgrid);
  }


  ///
  DisjointBoxLayout(const DisjointBoxLayout& a_input)
  {
    if(&a_input != this)
    {
      m_internals = a_input.m_internals;
    }
  }

  ///
  DisjointBoxLayout& operator=(const DisjointBoxLayout& a_input)
  {
    if(&a_input != this)
    {
      m_internals = a_input.m_internals;
    }
    return *this;
  }

  ///
  bool operator==(const DisjointBoxLayout& a_input) const
  {
    return (m_internals == a_input.m_internals);
  }

  ///
  void define(const Box& a_domain, const unsigned int& a_maxgrid)
  {
    PROTO_ASSERT(a_domain.coarsenable(a_maxgrid), "invalid dbl combo");

    m_internals = shared_ptr<localData>(new localData());

    m_internals->m_coarsenedDom = a_domain.coarsen(a_maxgrid);
    m_internals->m_maxgrid = a_maxgrid;

    //should probably do some sort of nearest neighbor walk
    unsigned int numboxes = m_internals->m_coarsenedDom.size();
    m_internals->m_procs.resize(numboxes);
    unsigned int boxesperproc = numboxes/(CH_numProc());
    for(unsigned int ibox = 0; ibox < numboxes; ibox++)
    {
      unsigned int boxproc = ibox/boxesperproc;
      m_internals->m_procs[ibox] = boxproc;
      unsigned int procid = CH_procID();
      if(boxproc == procid)
      {
        m_internals->m_localBoxes.push_back(ibox);
      }
    }

  }

  ///
  unsigned  int procID(unsigned int a_index) const
  {
    PROTO_ASSERT(m_internals,"trying to access undefined dbl procids");
    Point coarpt = m_internals->m_coarsenedDom[a_index];
    return m_internals->m_procs[a_index];
  }

  ///
  Box operator[](unsigned int a_index) const
  {
    PROTO_ASSERT(m_internals,"trying to access undefined dbl boxes");
    Point coarpt = m_internals->m_coarsenedDom[a_index];
    Box coarBox(coarpt, coarpt);
    Box retval = coarBox.refine(m_internals->m_maxgrid);
    return retval;
  }


  ///number of boxes in grid (over all procs)
  unsigned int size() const
  {
    return m_internals->m_coarsenedDom.size();
  }

  ///boxes in grid whose data lives on the current proc
  const vector<unsigned int>& localBoxes() const
  {
    PROTO_ASSERT(m_internals,"trying to access undefined dbl local boxes");
    return m_internals->m_localBoxes;
  }

};

///data over a disjointboxlayout with ghost cells
template <class T>
class LevelData
{

public:
  ///get to the data on a particular box.  this index is into m_data---the boxes on THIS processor.
  /**
     you can get a vector of these boxes by calling DisjointBoxLayout::localBoxes
   */
  T & operator[](unsigned int a_index)
  {
    
    PROTO_ASSERT(m_isDefined,"trying to access undefined leveldata");
    PROTO_ASSERT(a_index < m_data.size(),"bogus index sent to leveldata");
    
    return (*(m_data[a_index]));
  }

  ///
  LevelData()
  {
    m_isDefined = false;
  }


  ///
  LevelData(const DisjointBoxLayout& a_grids)
  {
    define(a_grids);
  }

  ///
  void define(const DisjointBoxLayout& a_grids, const Point& a_grow)
  {
    m_isDefined = true;
    m_grids = a_grids;
    const vector<unsigned int>& localBoxes = a_grids.localBoxes();
    m_data.resize(localBoxes.size());
    for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
    {
      int boxid   = localBoxes[ibox];
      Box dblbox  = a_grids[boxid];
      Box databox = dblbox.grow(a_grow);
      m_data[ibox] = shared_ptr<T>(new T(databox));
    }
    
  }

  ///return the number of boxes on THIS proc
  unsigned int size() const
  {
    return m_data.size();
  }

  void setToZero()
  {
    for(unsigned int ibox = 0; ibox < m_data.size(); ibox++)
    {
      m_data[ibox]->setVal(0);
    }
  }
private: 
  //in parallel, this is be the data on this proc
  vector<shared_ptr<T> >        m_data;
  DisjointBoxLayout             m_grids;
  bool                          m_isDefined;
};


///
inline void sync()
{
  #ifdef PROTO_CUDA
    {
      PR_TIME("device sync");
      protoDeviceSynchronize();
    }
#endif
}
/**/

void
doSomeForAlls(  LevelData< BoxData<double, NUMCOMPS> > & a_out,
                LevelData< BoxData<double, NUMCOMPS> > & a_low,
                LevelData< BoxData<double, NUMCOMPS> > & a_hig,
                const DisjointBoxLayout & a_dbl,
                const unsigned int      & a_numapplies,
                const unsigned int      & a_numstream,
                const unsigned int      & a_nmult)
{

  //remember this is just for timings
  vector<unsigned int> localBoxes = a_dbl.localBoxes();
  cout << "local boxes size  = " << localBoxes.size() << endl;
  for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
  {
    a_out[ibox].setVal(1.);
    a_hig[ibox].setVal(1.);
    a_low[ibox].setVal(1.);
  }
  vector<protoStream_t> streams(a_numstream);
  double gamma = 1.4;
  int idir = 0;
  for(unsigned int ibox = 0; ibox < a_numstream; ibox++)
  {
    protoStreamCreate(&streams[ibox]);
  }


  {
    PR_TIME("No_Z_increment");
    cout << "no z incr " << endl;
    {
      cout << "doing riemann problems " << endl;
      for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
      {
        for(unsigned int iapp = 0; iapp < a_numapplies; iapp++)
        {
          PR_TIME("riemann_on_level_multiStream");
          int istream = iapp%a_numstream;
          Box appBox       = a_dbl[localBoxes[ibox]];

          unsigned long long int count = (28 + a_nmult)*appBox.size();
          PR_FLOPS(count);
          cudaForallStruct(streams[istream], UpwindStruct(), appBox, a_out[ibox], a_low[ibox], a_hig[ibox], idir, gamma, a_nmult);

        }
      }
      sync();
    }

    {
      cout << "doing empty foralls " << endl;
      for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
      {
        for(unsigned int iapp = 0; iapp < a_numapplies; iapp++)
        {
          PR_TIME("do_nothing_on_level_multiStream");
          int istream = iapp%a_numstream;
          Box appBox       = a_dbl[localBoxes[ibox]];

          cudaForallStruct(streams[istream], DoNothingStruct()  , appBox, a_out[ibox], a_low[ibox], a_hig[ibox], idir, gamma, a_nmult);
        }
      }
      sync();
    }

  }

  {
    PR_TIME("Z_increment_Version");
    cout << "z incr version" << endl;
    {
      cout << "doing riemann problems " << endl;
      for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
      {
        for(unsigned int iapp = 0; iapp < a_numapplies; iapp++)
        {
          PR_TIME("riemann_on_level_multiStream");
          int istream = iapp%a_numstream;
          Box appBox       = a_dbl[localBoxes[ibox]];

          unsigned long long int count = (28 + a_nmult)*appBox.size();
          PR_FLOPS(count);
          cudaForallZincStruct(streams[istream], UpwindStruct(), appBox, a_out[ibox], a_low[ibox], a_hig[ibox], idir, gamma, a_nmult);

        }
      }
      sync();
    }

    {
      cout << "doing empty foralls " << endl;
      for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
      {
        for(unsigned int iapp = 0; iapp < a_numapplies; iapp++)
        {
          PR_TIME("do_nothing_on_level_multiStream");
          int istream = iapp%a_numstream;
          Box appBox       = a_dbl[localBoxes[ibox]];

          cudaForallZincStruct(streams[istream], DoNothingStruct()  , appBox, a_out[ibox], a_low[ibox], a_hig[ibox], idir, gamma, a_nmult);
        }
      }
      sync();
    }

  }


  for(unsigned int ibox = 0; ibox < a_numstream; ibox++)
  {
    protoStreamDestroy(streams[ibox]);
  }
}
/**/
int main(int argc, char* argv[])
{
  //have to do this to get a time table
  PR_TIMER_SETFILE("proto.time.table");
  unsigned int nx, niter, maxgrid,  nmult;
  parseCommandLine(nmult, nx, maxgrid, niter,  argc, argv);

  Point lo = Point::Zeros();
  Point hi = Point::Ones(nx - 1);
  Box domain(lo, hi);
  
  DisjointBoxLayout dbl(domain, maxgrid);
  LevelData<BoxData<double, NUMCOMPS> > out, hig, low;

  {
    
    PR_TIME("data definition");

    out.define(dbl, Point::Zeros());
    hig.define(dbl, Point::Zeros());
    low.define(dbl, Point::Zeros());

  }

  {
    PR_TIME("1_STREAM");
    int nstream = 1;
    cout << "running test with " << nstream << " stream(s)" << endl;
    doSomeForAlls(out, hig, low, dbl, niter, nstream, nmult);
  }
  {
    PR_TIME("2_STREAMS");
    int nstream = 2;
    cout << "running test with " << nstream << " stream(s)" << endl;
    doSomeForAlls(out, hig, low, dbl, niter, nstream, nmult);
  }

  {
    PR_TIME("4_STREAMS");
    int nstream = 4;
    cout << "running test with " << nstream << " stream(s)" << endl;
    doSomeForAlls(out, hig, low, dbl, niter, nstream, nmult);
  }

  {
    PR_TIME("8_STREAMS");
    int nstream = 8;
    cout << "running test with " << nstream << " stream(s)" << endl;
    doSomeForAlls(out, hig, low, dbl, niter, nstream, nmult);
  }


  {
    PR_TIME("16_STREAMS");
    int nstream = 16;
    cout << "running test with " << nstream << " stream(s)" << endl;
    doSomeForAlls(out, hig, low, dbl, niter, nstream, nmult);
  }

  {
    PR_TIME("32_STREAMS");
    int nstream = 32;
    cout << "running test with " << nstream << " stream(s)" << endl;
    doSomeForAlls(out, hig, low, dbl, niter, nstream, nmult);
  } 


  PR_TIMER_REPORT();

}  
