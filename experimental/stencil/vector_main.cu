#include "hip/hip_runtime.h"
/*
  Copyright Marcin Krotkiewski, University of Oslo, 2012


  updated by Brian Van Straalen 2018
   updated for the loss of cutil.h
   updated to use 6 streams of execution
   updated to use std::chrono

  compile command
  >nvcc -std=c++11 -O3 vector_main.cu -o vector_main.exe

  options:   -nx
             -ny
             -nz
             -nstreams
             -nbox
             -iter
             -texsize
             -pitch
             -pitchy
             -routine


*/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <malloc.h>
#include <sys/time.h>
#include <time.h>
#include <errno.h>
#include <chrono>
#include <algorithm>
#include <iostream>
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#define HERE fprintf(stderr, "HERE %d\n", __LINE__)
#define MSINGLE
//#undef MSINGLE
#ifdef MSINGLE
typedef float mfloat;
#else
typedef double mfloat;
#endif


#define cutilSafeCall(err)     __cudaSafeCall   (err, __FILE__, __LINE__)
#define cutilCheckError(err)   __cutilCheckError(err, __FILE__, __LINE__)
inline void __cudaSafeCall(hipError_t err,
                           const char *file, const int line){
  if(hipSuccess != err) {
    printf("%s(%i) : cutilSafeCall() Runtime API error : %s.\n",
           file, line, hipGetErrorString(err) );
    exit(-1);
  }
}
inline void __cutilCheckError( bool err, const char *file, const int line )
{
    if( true != err) {
        fprintf(stderr, "CUTIL CUDA error in file <%s>, line %i.\n",
                file, line);
        exit(-1);
    }
}
mfloat h_kernel_3c_all[3*3*3] = {-1./12, -1./6, -1./12,
				 -1./6 ,  0. , -1./6,
				 -1./12, -1./6, -1./12,
			 
				 -1./6 , 0., -1./6,
				 0., 2.+2.0/3.0, 0.,
				 -1./6, 0., -1./6,
			 
				 -1./12, -1./6, -1./12,
				 -1./6 ,  0. , -1./6,
				 -1./12, -1./6, -1./12};

__device__ __constant__ mfloat d_kernel_3c[3*3*3];


#ifdef MSINGLE
texture<float, 1, hipReadModeElementType> texData1D;
#else
texture<int2 , 1, hipReadModeElementType> texData1D;
#endif

hipChannelFormatDesc floatTex;
hipExtent gridExtent;

hipArray *cu_array;
//hipPitchedPtr p_T1, p_T2;
//mfloat *d_T1, *d_T2;
//mfloat *h_T1, *h_T2;


extern "C"{
#include "kernels.cu"
}

__inline__ mfloat host_convolution_3x3(const mfloat *kernel, const mfloat *data,
				       const int tx, const int ty, const int bx,
				       int nx, int ny, int nz)
{
  // periodic boundaries in X-Y
  int txm = (tx-1+nx)%(nx);
  int txp = (tx+1)%(nx);
  int tym = (ty-1+ny)%(ny);
  int typ = (ty+1)%(ny);

  return 
    kernel[0]*data[txm + tym*bx] +
    kernel[1]*data[tx  + tym*bx] +
    kernel[2]*data[txp + tym*bx] +

    kernel[3]*data[txm + ty*bx] +
    kernel[4]*data[tx  + ty*bx] +
    kernel[5]*data[txp + ty*bx] +

    kernel[6]*data[txm + typ*bx] +
    kernel[7]*data[tx  + typ*bx] +
    kernel[8]*data[txp + typ*bx] ;
}


void host_convolution(mfloat *out, const mfloat *in, int nx, int ny, int nz, int pitch, int pitchy, mfloat *kernel)
{
  mfloat temp;
  
  for(int k=1; k<nz-1; k++){
    for(int j=0; j<ny; j++){
      for(int i=0; i<nx; i++){
	temp  =  host_convolution_3x3(kernel   , in + (k-1)*pitch*pitchy, i, j, pitch, nx, ny, nz);
	temp +=  host_convolution_3x3(kernel+9 , in + (k-0)*pitch*pitchy, i, j, pitch, nx, ny, nz);
	temp +=  host_convolution_3x3(kernel+18, in + (k+1)*pitch*pitchy, i, j, pitch, nx, ny, nz);
	out[k*pitch*pitchy + j*pitch + i] = temp;
      }
    }
  }
}


void copy_cube_simple(void *d, void *s, int nx, int ny, int nz, int kind)
{
  switch(kind){
  case hipMemcpyHostToDevice:
    cutilSafeCall(hipMemcpy(d, s, nx*ny*nz*sizeof(mfloat), hipMemcpyHostToDevice));
    break;
  case hipMemcpyDeviceToDevice:
    cutilSafeCall(hipMemcpy(d, s, nx*ny*nz*sizeof(mfloat), hipMemcpyDeviceToDevice));
    break;
  case hipMemcpyDeviceToHost:
    cutilSafeCall(hipMemcpy(d, s, nx*ny*nz*sizeof(mfloat), hipMemcpyDeviceToHost));
    break;
  }
}

 using namespace std::chrono;


void ctoc(high_resolution_clock::time_point timer, uint iters, float unit_mem, int nrw_center, int nro_halo, int thrdim_x, int thrdim_y, int nx, int ny, int nz)
{  
  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  duration<double> time_span = duration_cast<duration<double>>(t2-timer);
  double fps = ((double)iters) / (time_span.count());
  double halo_overhead = (double)(2*thrdim_x + 2*(thrdim_y+2)*32/sizeof(mfloat))/(double)(thrdim_x*thrdim_y);
  double effmembwd = (nrw_center+nro_halo)*unit_mem / (time_span.count()/ (double)iters);
  double hwmembwd  = (nrw_center+nro_halo+halo_overhead)*unit_mem / (time_span.count()/ (double)iters);
  double ptsthrough = (double)nx*ny*nz*iters/(double)(time_span.count());
  fprintf(stderr, "(%d, %d, %d): (TX, TY) = (%d, %d), fps %e, time %e, pts/s %e, effmembwd GB/s %3.1e, overhead %3.3e hwmembwd (GB/s) %3.3e)\n", 
	  nx, ny, nz, thrdim_x, thrdim_y, fps, time_span.count(), ptsthrough, effmembwd/1e9, halo_overhead, hwmembwd/1e9);
}


void compute_difference(void *ptr, mfloat *h_T1, mfloat *h_T2, int nx, int ny, int nz, int pitch, int pitchy, int thrdim_x, int thrdim_y, float iters)
{
  double temp = 0;

  bzero(h_T1, sizeof(mfloat)*pitch*pitchy*nz);
  if(ptr){
    copy_cube_simple(h_T1, ptr, pitch, pitchy, nz, hipMemcpyDeviceToHost);
  }

  for(int k=0; k<nz; k++){
    for(int j=0; j<ny; j++){
      for(int i=0; i<nx; i++){
	temp  =  std::max(temp, (double)fabs(h_T1[k*pitch*pitchy + j*pitch + i] - h_T2[k*pitch*pitchy + j*pitch + i]));
      }
    }
  }

  printf("validation, CPU vs GPU: %e\n", temp);
} 


dim3 get_grid(dim3 block, int nx, int ny, int nz, int thrdim_x, int thrdim_y)
{
  int modx = nx%thrdim_x;
  int mody = ny%thrdim_y;
  
  dim3 grid(nx/block.x, ny/block.y, 1);
  if(modx){
    grid.x++;
  }
  if(mody){
    grid.y++;
  }
  return grid;
}

void GetCmdLineArgumenti(int argc, const char** argv, const char* name, int* rtn)
{
  size_t len = strlen(name);
  for(int i=1; i<argc; i+=2)
    {
      if(strcmp(argv[i]+1,name) ==0)
        {
         *rtn = atoi(argv[i+1]);
         std::cout<<name<<"="<<" "<<*rtn<<std::endl;
          break;
        }
    }
}

int bigTest(int argc, char*argv[])
{

  using std::vector;
  int device = 0;
  int nx = 64;
  int ny = 64;
  int nz = 64;
  int iters = 10;

  int routine = 1, thrdim_x = 32, thrdim_y = 6;
  int texsize = 22;
  int nstream = 8;
  int nbox = 128;
  /* -------------------- */
  /* command-line parameters */
  /* -------------------- */
  GetCmdLineArgumenti(argc, (const char**)argv, "nx", &nx);
  ny = nx;
  nz = nx;
  GetCmdLineArgumenti(argc, (const char**)argv, "ny", &ny);
  GetCmdLineArgumenti(argc, (const char**)argv, "nz", &nz);
  GetCmdLineArgumenti(argc, (const char**)argv, "nbox", &nbox);
  GetCmdLineArgumenti(argc, (const char**)argv, "nstream", &nstream);
  GetCmdLineArgumenti(argc, (const char**)argv, "routine", &routine);
  GetCmdLineArgumenti(argc, (const char**)argv, "device", &device);
  GetCmdLineArgumenti(argc, (const char**)argv, "iters", &iters);
  vector<hipStream_t> streams(nstream);
  for(int istream = 0; istream < nstream; istream++)
  {
    hipStreamCreate(&streams[istream]);
  }

  /* choose device */
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  hipSetDevice(device);
  if(strstr(deviceProp.name, "1060")){
    texsize = 22;
  } else {
    texsize = 24;
  }
  GetCmdLineArgumenti(argc, (const char**)argv, "texsize", &texsize);
  printf("using device %s, using linear texture size: 2^%d elements\n", deviceProp.name, texsize);

  int pitch  = nx;
  int pitchy = ny;

  /* special case - much worse performance with default pitch on tesla1060 */
  if(nx==352) pitch = nx + 64;

  GetCmdLineArgumenti(argc, (const char**)argv, "pitch", &pitch);
  GetCmdLineArgumenti(argc, (const char**)argv, "pitchy", &pitchy);


  /* -------------------- */
  /* Initialization */
  /* -------------------- */

  /* initialize texture */
#ifdef MSINGLE
  floatTex = hipCreateChannelDesc<float>();
#else
  floatTex = hipCreateChannelDesc<int2>();
#endif

  /* allocate alligned 3D data on the GPU */
  gridExtent = make_hipExtent(pitch*sizeof(mfloat), pitchy, nz);

  std::cout<< "grid extent depth = " << gridExtent.depth << ", height = "<< gridExtent.height << ", width = " << gridExtent.width << std::endl;
  printf("nx=%d,ny=%d,nz=%d\n", nx, ny, nz);

  printf("nbox = %d, nstream = %d, niter = %d \n", nbox, nstream, iters);
  vector<hipPitchedPtr> vec_p_T1(nbox);
  vector<hipPitchedPtr> vec_p_T2(nbox);
  vector<mfloat*> vec_h_T1(nbox);
  vector<mfloat*> vec_h_T2(nbox);
  vector<mfloat*> vec_d_T1(nbox);
  vector<mfloat*> vec_d_T2(nbox);

  for(int ibox = 0; ibox < nbox; ibox++)
  {
 
    cutilSafeCall(hipMalloc3D(&(vec_p_T1[ibox]), gridExtent));
    cutilSafeCall(hipMalloc3D(&(vec_p_T2[ibox]), gridExtent));

    vec_d_T1[ibox]  = (mfloat*)(vec_p_T1[ibox].ptr);
    vec_d_T2[ibox]  = (mfloat*)(vec_p_T2[ibox].ptr);
  }

  //set memory and allocate host data
  for(int ibox = 0; ibox < nbox; ibox++)
  {
 

    mfloat* h_T1 = vec_h_T1[ibox];
    mfloat* h_T2 = vec_h_T2[ibox];
    mfloat* d_T1 = vec_d_T1[ibox];
    mfloat* d_T2 = vec_d_T2[ibox];

    pitch = vec_p_T1[ibox].pitch/sizeof(mfloat);

    cutilSafeCall(hipMemset(vec_d_T1[ibox], 0, pitch*pitchy*nz*sizeof(mfloat)));
    cutilSafeCall(hipMemset(vec_d_T2[ibox], 0, pitch*pitchy*nz*sizeof(mfloat)));

      /* allocate and initialize host data */
    h_T1 = (mfloat*)calloc(pitch*pitchy*nz, sizeof(mfloat));
    h_T2 = (mfloat*)calloc(pitch*pitchy*nz, sizeof(mfloat)); 

    srand(1);
    for(long i=0; i<pitch*pitchy*nz; i++) 
      h_T1[i] = 1.0 - 2.0*(double)rand()/RAND_MAX;

    /* copy data to the GPU */
    copy_cube_simple(d_T1, h_T1, pitch, pitchy, nz, hipMemcpyHostToDevice);


  }
  /* copy stencil to the GPU */
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_3c), h_kernel_3c_all,
                                   sizeof(mfloat)*27, 0, hipMemcpyHostToDevice));

  /* -------------------- */
  /* performance tests    */
  /* -------------------- */
  
  
  high_resolution_clock::time_point time_start = high_resolution_clock::now(); 

  for(int ibox = 0; ibox < nbox; ibox++)
  {
 
    int istream = ibox%nstream;

    mfloat* h_T1 = vec_h_T1[ibox];
    mfloat* h_T2 = vec_h_T2[ibox];
    mfloat* d_T1 = vec_d_T1[ibox];
    mfloat* d_T2 = vec_d_T2[ibox];

    pitch = vec_p_T1[ibox].pitch/sizeof(mfloat);
    high_resolution_clock::time_point tstart = high_resolution_clock::now();
    for(int it=0; it<iters; it++)
    {

      dim3 block(thrdim_x, thrdim_y, 1);
      dim3 grid = get_grid(block, nx, ny, nz, thrdim_x, thrdim_y);
    
      int kstep  = std::min((1<<texsize)/(pitch*pitchy), nz);
      //printf("kstep %d\n", kstep);
    
      int kstart = 1;
      int kstop;
      size_t texoffset;
 
      while(1)
      {
      
        kstop = std::min(kstart+kstep-2, nz-1);
        //printf("kstart %d, kstop %d\n", kstart, kstop);
        cutilSafeCall(hipBindTexture(&texoffset, &texData1D, d_T1+(kstart-1)*pitch*pitchy, 
                                      &floatTex, pitch*pitchy*kstep*sizeof(mfloat)));
 
        texoffset = texoffset/sizeof(mfloat);
      
        if(routine==1)
          stencil27_symm_exp_tex<<<grid, block, 2*(block.x)*(block.y)*sizeof(mfloat),streams[istream]>>>
            (d_T2, 0, 0, nx, ny, nz, pitch, pitchy, texoffset, kstart, kstop);
        else if(routine==2)
          stencil27_symm_exp_tex_prefetch<<<grid, block, 2*(block.x)*(block.y)*sizeof(mfloat),streams[istream]>>>
            (d_T2, 0, 0, nx, ny, nz, pitch, pitchy, texoffset, kstart, kstop);
        else if(routine==3)
          stencil27_symm_exp_tex_new<<<grid, block, 2*(block.x)*(block.y)*sizeof(mfloat),streams[istream]>>>
            (d_T2, 0, 0, nx, ny, nz, pitch, pitchy, texoffset, kstart, kstop);
        else
          stencil27_symm_exp_tex_prefetch_new<<<grid, block, 2*(block.x)*(block.y)*sizeof(mfloat),streams[istream]>>>
            (d_T2, 0, 0, nx, ny, nz, pitch, pitchy, texoffset, kstart, kstop);
      
        kstart = kstop;
        if(kstart>=nz-1) break;
      }
    }
    /* finalize */
    hipDeviceSynchronize();
    //unsigned long long int numflops = 2*iters*27*nx*ny*nz;
 
  }
  high_resolution_clock::time_point time_end = high_resolution_clock::now(); 
  duration<double> time_span = duration_cast<duration<double>>(time_end-  time_start);
  double microseconds = 1.0e6*(time_span.count());
  long long nptsperbox = nx*ny*nz;
  long long flops =  2*iters*27*(nptsperbox)*nbox;
  double mega_flop_rate = flops/microseconds;
  std::cout << "nx = "<< nx << ",ny= " << ny << ",nz= " << nz << ",nbox=" << nbox << ",iters = " << iters << std::endl;
  std::cout << std::scientific << "time = " << microseconds << "mu s, num ops= " << flops << ", flop rate = " << mega_flop_rate << "MFlops"  << std::endl;
//  ctoc(timer, iters, nbox*nx*ny*nz*sizeof(mfloat), 1, 1, thrdim_x, thrdim_y, nx, ny, nz);   
  
  /* perform computations on host */
//
//  bzero(h_T2, sizeof(mfloat)*pitch*pitchy*nz); 
//  host_convolution(h_T2, h_T1, nx, ny, nz, pitch, pitchy, h_kernel_3c_all);
//
//  /* compute difference in the results */
//  compute_difference(d_T2, h_T1, h_T2, nx, ny, nz, pitch, pitchy, thrdim_x, thrdim_y, iters);


  for(int istream = 0; istream < nstream; istream++)
  {
    hipStreamDestroy(streams[istream]);
  }
  return 0;
}


int main(int argc, char*argv[])
{

  
  int retval = bigTest(argc, argv);



  return retval;
}
