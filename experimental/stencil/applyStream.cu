#include <cstdio>
#include <cstring>
#include <cassert>
#include <cmath>

#include <vector>
#include <memory>

#include <iostream>
#include <fstream>
#include <sstream>

#define DIM 2
#include "../../include/Proto.H"
using std::cout;
using std::endl;
using std::vector;
using std::shared_ptr;
using namespace Proto;
/**/
void
parseCommandLine(unsigned int & a_nx, unsigned int& a_maxgrid, unsigned int & a_numapplies, int argc, char* argv[])
{
  //defaults
  a_nx = 16;
  a_numapplies = 1;
  a_maxgrid = 4;
  cout << "kernel timings of various laplacians" << endl;
  cout << "usage:  " << argv[0] << " -n nx[default:8] -a num_iterations[default:1]" << endl;
  for(int iarg = 0; iarg < argc-1; iarg++)
  {
    if(strcmp(argv[iarg],"-n") == 0)
    {
      a_nx = atoi(argv[iarg+1]);
    }
    else if(strcmp(argv[iarg], "-a") == 0)
    {
      a_numapplies = atoi(argv[iarg+1]);
    }
    else if(strcmp(argv[iarg], "-m") == 0)
    {
      a_maxgrid = atoi(argv[iarg+1]);
    }
  }
}


///proxies for Chombo-style SPMD functions
unsigned int numProc()
{
  return 1;
}

unsigned int procID()
{
  return 0;
}

///boxes that cover a domain box
class DisjointBoxLayout
{
private:
  class localData
  {
  public:
    localData() {;}
    Box                   m_coarsenedDom;
    vector<unsigned int>  m_procs;
    vector<unsigned int>  m_localBoxes;
    unsigned int          m_maxgrid;
  };

  //this is to make this a ref-counted object
  shared_ptr<localData> m_internals;


public:
  ///
  DisjointBoxLayout(const Box& a_domain, const unsigned int& a_maxgrid)
  {
    define(a_domain, a_maxgrid);
  }


  ///
  DisjointBoxLayout(const DisjointBoxLayout& a_input)
  {
    if(&a_input != this)
    {
      m_internals = a_input.internals;
    }
  }

  ///
  DisjointBoxLayout& operator=(const DisjointBoxLayout& a_input)
  {
    if(&a_input != this)
    {
      m_internals = a_input.internals;
    }
    return *this;
  }

  ///
  bool operator==(const DisjointBoxLayout& a_input) const
  {
    return (m_internals == a_input.m_internals);
  }

  ///
  void define(const Box& a_domain, const unsigned int& a_maxgrid)
  {
    PROTO_ASSERT(a_domain.coarsenable(a_maxgrid), "invalid dbl combo");

    m_internals = shared_ptr<localData>(new localData());

    m_internals->m_coarsenedDom = a_domain.coarsen(a_maxgrid);
    m_internals->m_maxgrid = a_maxgrid;

    //should probably do some sort of nearest neighbor walk
    unsigned int numboxes = m_internals->m_coarsenedDom.size()
    m_internals->m_procs.resize(numboxes);
    unsigned int boxesperproc = numboxes/numProc();
    m_internals->m_numBoxesThisProc = 0;
    for(unsigned int ibox = 0; ibox > numboxes; ibox++)
    {
      unsigned int boxproc = ibox/boxesperproc;
      m_internals->m_procs[ibox] = boxproc;
      if(boxproc == procID())
      {
        m_internals->m_localBoxes.push_back(ibox);
      }
    }

  }

  ///
  unsigned  int procID(int a_index) const
  {
    PROTO_ASSERT(m_internals,"trying to access undefined dbl procids");
    Point coarpt = m_internals->m_coarsenedDom.index(a_index);
    return m_internals->m_procs[a_index];
  }

  ///
  Box operator[](int a_index) const
  {
    PROTO_ASSERT(m_internals,"trying to access undefined dbl boxes");
    Point coarpt = m_internals->m_coarsenedDom.index(a_index);
    Box retval(coarpt, coarpt);
    retval.refine(m_internals->m_maxgrid);
    return retval;
  }


  ///number of boxes in grid (over all procs)
  unsigned int size() const
  {
    return m_internals->m_coarsenedDom.size();
  }

  ///boxes in grid whose data lives on the current proc
  const vector<unsigned int>& localBoxes() const
  {
    PROTO_ASSERT(m_internals->m_isDefined,"trying to access undefined dbl size");
    return m_internals->m_localBoxes;
  }

};

///data over a disjointboxlayout with ghost cells
template <class T>
class LevelData
{

  ///get to the data on a particular box.  this index is into m_data---the boxes on THIS processor.
  /**
     you can get a vector of these boxes by calling DisjointBoxLayout::localBoxes
   */
  T & operator[](unsigned int a_index)
  {
    
    PROTO_ASSERT(m_isDefined,"trying to access undefined leveldata");
    PROTO_ASSERT(a_index < m_data.size(),"bogus index sent to leveldata");
    
    return (*(m_data[a_index]));
  }

  ///
  LevelData()
  {
    m_isDefined = false;
  }


  ///
  LevelData(const DisjointBoxLayout& a_grids)
  {
    define(a_grids);
  }

  ///
  void define(const DisjointBoxLayout& a_grids, const Point& a_grow)
  {
    m_isDefined = true;
    m_grids = a_grids;
    const vector<unsigned int>& localBoxes = a_grids.localBoxes();
    m_data.resize(localBoxes.size());
    for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
    {
      int boxid   = localBoxes[ibox];
      Box dblbox  = a_grids[boxid];
      Box databox = dblbox.grow(a_grow);
      m_data[ibox] = shared_ptr<T>(new T(databox));
    }
    
  }

  ///return the number of boxes on THIS proc
  unsigned int size() const
  {
    return m_data.size();
  }

  void setVal(const T& a_val)
  {
    for(unsigned int ibox = 0; ibox < m_data.size(); ibox++)
    {
      m_data[ibox].setVal(a_val);
    }
  }
private: 
  //in parallel, this is be the data on this proc
  vector<shared_ptr<T> >        m_data;
  DisjointBoxLayout             m_grids;
  bool                          m_isDefined();
};


///
inline void sync()
{
  #ifdef PROTO_CUDA
    {
      PR_TIME("device sync");
      hipDeviceSynchronize();
    }
#endif
}
/**/

template <class T> void
applyLaplacians(LevelData< BoxData<T> > & a_phi,
                LevelData< BoxData<T> > & a_lap,
                const DisjointBoxLayout & a_dbl,
                const unsigned int      & a_numapplies)
{

  PR_TIME("applyLaplacians");
#if DIM==2
  Stencil<T> lapSten = Stencil<T>::Laplacian_9();
#else 
  Stencil<T> lapSten = Stencil<T>::Laplacian_27();
#endif

  //remember this is just for timings
  a_phi.setVal(0.);
  a_lap.setVal(0.);
  vector<unsigned int> localBoxes = a_dbl.localBoxes();
  vector<hipStream_t> streams(localBoxes.size());
  for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
  {
    hipStreamCreate(&streams[ibox]);
  }
  {
    PR_TIME("applyLaplacianStencilOnLevel");
    for(unsigned int iapp = 0; iapp < a_numapplies; iapp++)
    {
      for(unsigned int ibox = 0; ibox < localBoxes.size(); ibox++)
      {
        Box appBox       = a_dbl[localBoxes[ibox]];
        int streamNumber = ibox;  //perhaps this needs to be thought out better
        lapSten.cudaApplyStream(phi[ibox], lap[ibox], appBox, true, 1.0, streams[ibox]);
      }
    }
    sync();
  }

//  Stencil<T> emptySten;
}
/**/
int main(int argc, char* argv[])
{
  //have to do this to get a time table
  PR_TIMER_SETFILE("proto.time.table");
  unsigned int nx, niter, maxgrid;
  parseCommandLine(nx, maxgrid, niter, argc, argv);

  Point lo = Point::Zeros();
  Point hi = Point::Ones(nx - 1);
  Box domain(lo, hi);
  
  DisjointBoxLayout dbl(domain, maxgrid);
  LevelData<BoxData<double> > phid, lapd;
  LevelData<BoxData<float>  > phif, lapf;

  {
    
    PR_TIME("data definition");

    Point ghostPt = Point::Ones();
    Point noGhost = Point::Zeros();
    phid.define(dbl, ghostPt);
    phif.define(dbl, ghostPt);
    lapd.define(dbl, noGhost);
    lapf.define(dbl, noGhost);

  }
  {
    PR_TIME("SINGLE_precision_laplacian");
    applyLaplacians<float >(phif, lapf, dbl, numapplies);
  }

  {
    PR_TIME("DOUBLE_precision_laplacian");
    applyLaplacians<float >(phid, lapd, dbl, numapplies);
  }


  PR_TIMER_REPORT();

}  
