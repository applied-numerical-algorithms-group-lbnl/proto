#include "hip/hip_runtime.h"
/*
  Copyright Marcin Krotkiewski, University of Oslo, 2012
*/

#define stencil_3x3(c0, c1, c2, shm, tx, ty, bx)			\
  c0*((shm)[tx+0+(ty+0)*bx]) +						\
  c1*((shm)[tx-1+(ty+0)*bx]  + (shm)[tx+1+(ty+0)*bx] + (shm)[tx+0+(ty-1)*bx] + (shm)[tx+0+(ty+1)*bx]) + \
  c2*((shm)[tx-1+(ty-1)*bx]  + (shm)[tx+1+(ty-1)*bx] + (shm)[tx-1+(ty+1)*bx] + (shm)[tx+1+(ty+1)*bx])
  

__device__ inline mfloat stencil_3x3_function(mfloat c0, mfloat c1, mfloat c2, mfloat* shm,
                                uint tx, uint ty, uint bx)
{
  mfloat rtn = 0;
  rtn+=  c0*((shm)[tx+0+(ty+0)*bx]);
  rtn+=  c1*((shm)[tx-1+(ty+0)*bx]  + (shm)[tx+1+(ty+0)*bx] + (shm)[tx+0+(ty-1)*bx] + (shm)[tx+0+(ty+1)*bx]);
  rtn += c2*((shm)[tx-1+(ty-1)*bx]  + (shm)[tx+1+(ty-1)*bx] + (shm)[tx-1+(ty+1)*bx] + (shm)[tx+1+(ty+1)*bx]);
  return rtn;
}
  
  
#define stencil_3x3_reg(c0, c1, c2)					\
  c0*r5 +								\
  c1*(r2+r4+r6+r8) +							\
  c2*(r1+r3+r7+r9)

#define push_regs_exp(shm, bx)			\
  {						\
    r1=(shm)[tx-1+(ty-1)*bx];			\
    r2=(shm)[tx+0+(ty-1)*bx];			\
    r3=(shm)[tx+1+(ty-1)*bx];			\
						\
    r4=(shm)[tx-1+(ty+0)*bx];			\
    r5=(shm)[tx+0+(ty+0)*bx];			\
    r6=(shm)[tx+1+(ty+0)*bx];			\
	    					\
    r7=(shm)[tx-1+(ty+1)*bx];			\
    r8=(shm)[tx+0+(ty+1)*bx];			\
    r9=(shm)[tx+1+(ty+1)*bx];			\
  }						\

namespace cg = cooperative_groups;

__global__ void stencil27_symm_exp(mfloat *in, mfloat *out, 
				       uint dimx, uint dimy, uint dimz, 
				       uint kstart, uint kend)
{
  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const  int ix = blockIdx.x*blockDim.x + threadIdx.x;	
  const  int iy = blockIdx.y*blockDim.y + threadIdx.y;

  const uint ti = threadIdx.y*blockDim.x + threadIdx.x;
  const uint pad = 32/sizeof(mfloat); // halos to left & right of interior require 32 byte memory transaction
  const uint bx= blockDim.x+2*pad;
  const uint txe= ti%bx; // this thread's block-relative x-axis index for first read
  const uint tye= ti/bx; // this thread's block-relative y-axis index for first read
  const uint txe2= (ti+blockDim.x*blockDim.y)%bx; // because of halos, each thread reads two values
  const uint tye2= (ti+blockDim.x*blockDim.y)/bx;
  int  ixe= blockIdx.x*blockDim.x + txe - pad; // this thread's global x-axis index for first read
  int  iye= blockIdx.y*blockDim.y + tye - 1;
  int  ixe2= blockIdx.x*blockDim.x + txe2 - pad;
  int  iye2= blockIdx.y*blockDim.y + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(ixe2<0)      ixe2 += dimx;
  if(ixe2>dimx-1) ixe2 -= dimx;

  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];
  uint i1, i2;

  cg::thread_block block = cg::this_thread_block();
  extern __shared__ mfloat shm[];			

  i1 = ixe+iye*dimx;
  i2 = ixe2+iye2*dimy;

  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  block.sync();
   t1 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, bx);
  block.sync();

  i1 += dimx*dimy;
  i2 += dimx*dimy;

  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  block.sync();
  t2 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, bx);
  t1+= stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, bx);
  block.sync();

  for(uint kk=kstart; kk<kend; kk++){

    block.sync();

    i1 += dimx*dimy;
    i2 += dimx*dimy;

    shm[txe +tye *bx] = in[i1];
    shm[txe2+tye2*bx] = in[i2];

    block.sync();
    t3 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, bx);

    out[ix + iy*dimx + kk*dimx*dimy] = t1 + t3;
    t1 = t2 + stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, bx);
    t2 = t3;
  }
}


__global__ void stencil27_symm_exp_prefetch(mfloat *out, mfloat a, mfloat b,
						uint dimx, uint dimy, uint dimz,
                                                uint pitch, uint pitchy, mfloat* in, 
						uint kstart, uint kend)
{
  mfloat r1, r2, r3, r4, r5, r6, r7, r8, r9;

  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const  int ix = blockIdx.x*blockDim.x + threadIdx.x;	
  const  int iy = blockIdx.y*blockDim.y + threadIdx.y;

  const uint ti = threadIdx.y*blockDim.x + threadIdx.x;
  const uint pad = 32/sizeof(mfloat);
  const uint bx= blockDim.x+2*pad;
  const uint txe= ti%bx;
  const uint tye= ti/bx;
  const uint txe2= (ti+blockDim.x*blockDim.y)%bx;
  const uint tye2= (ti+blockDim.x*blockDim.y)/bx;
  int  ixe= blockIdx.x*blockDim.x + txe - pad;
  int  iye= blockIdx.y*blockDim.y + tye - 1;
  int  ixe2= blockIdx.x*blockDim.x + txe2 - pad;
  int  iye2= blockIdx.y*blockDim.y + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(ixe2<0)      ixe2 += dimx;
  if(ixe2>dimx-1) ixe2 -= dimx;

  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];

  uint i1, i2;

  cg::thread_block block = cg::this_thread_block();						
  extern __shared__ mfloat shm[];

  i1 = ixe+iye*pitch;
  i2 = ixe2+iye2*pitch;
  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  block.sync();  
  push_regs_exp(shm+pad+bx, bx); // pad+bx accounts for halos on top and to left of interior's start 
  block.sync();

  i1 += pitch*pitchy;
  i2 += pitch*pitchy;

  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  t1 = stencil_3x3_reg(C1, C2, C3);

  block.sync();  
  push_regs_exp(shm+pad+bx, bx);  
  block.sync();

  i1 += pitch*pitchy;
  i2 += pitch*pitchy;

  shm[txe +tye *bx] = in[i1];
  shm[txe2+tye2*bx] = in[i2];

  t2 = stencil_3x3_reg(C1, C2, C3);
  t1+= stencil_3x3_reg(C0, C1, C2);

  for(uint kk=kstart; kk<kend-1; kk++){

    block.sync();  
    push_regs_exp(shm+pad+bx, bx);  
    block.sync();

    i1 += pitch*pitchy;
    i2 += pitch*pitchy;

    shm[txe +tye *bx] = in[i1];
    shm[txe2+tye2*bx] = in[i2];

    t3 = stencil_3x3_reg(C1, C2, C3);

    out[ix + iy*pitch + kk*pitch*pitchy] = t1 + t3;
    t1 = t2 + stencil_3x3_reg(C0, C1, C2);
    t2 = t3;

  }

  block.sync();  
  push_regs_exp(shm+pad+bx, bx);  
  block.sync();

  out[ix + iy*pitch + (kend-1)*pitch*pitchy] = t1 + stencil_3x3_reg(C1, C2, C3);
}


__global__ void stencil27_symm_exp_new(mfloat *out, mfloat a, mfloat b,
					   uint dimx, uint dimy, uint dimz,
                                           uint pitch, uint pitchy, mfloat* in, 
					   uint kstart, uint kend)
{
  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const uint ix = blockIdx.x*32 + threadIdx.x; // 32 = blockDim.x
  const uint iy = blockIdx.y*6 + threadIdx.y; // 6 = blockDim.y
  const uint ti = threadIdx.y*32 + threadIdx.x;
  const uint pad = 32/sizeof(mfloat);
  const uint width = 32+2*pad; // width of slice, including halos
  const uint tye= ti/width;
  const uint txe= ti-tye*width;
  const uint tye2=tye+4; // including halos, slice has 8 rows, so tye2 is 4 rows below tye

  int  ixe = blockIdx.x*32 + txe  - pad;
  int  iye = blockIdx.y*6  + tye  - 1;
  int  iye2= blockIdx.y*6  + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  uint i1, i2;
  
  i1 = ixe+iye*pitch ;
  i2 = ixe+iye2*pitch ;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];

  cg::thread_block block = cg::this_thread_block();
  extern __shared__ mfloat shm[];			

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  block.sync();
  t1 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, width);
  block.sync();

  i1 += pitch*pitchy;
  i2 += pitch*pitchy;

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  block.sync();
  t2 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, width);
  t1+= stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, width);
  block.sync();

  for(uint kk=kstart; kk<kend; kk++){

    block.sync();

    i1 += pitch*pitchy;
    i2 += pitch*pitchy;

    shm[txe +tye *width] = in[i1];
    shm[txe+tye2*width] = in[i2];

    block.sync();
    t3 = stencil_3x3(C1, C2, C3, shm, tx+pad, ty+1, width);

    out[ix + iy*pitch + kk*pitch*pitchy] = t1 + t3;
    t1 = t2 + stencil_3x3(C0, C1, C2, shm, tx+pad, ty+1, width);
    t2 = t3;
  }
}


__global__ void stencil27_symm_exp_prefetch_new(mfloat *out, mfloat a, mfloat b,
						    uint dimx, uint dimy, uint dimz,
                                                    uint pitch, uint pitchy, mfloat* in, 
						    uint kstart, uint kend)
{
  mfloat r1, r2, r3, r4, r5, r6, r7, r8, r9;

  const uint tx = threadIdx.x;
  const uint ty = threadIdx.y;
  const uint ix = blockIdx.x*32 + threadIdx.x;
  const uint iy = blockIdx.y*6  + threadIdx.y;
  const uint ti = threadIdx.y*32 + threadIdx.x;
  const uint pad= 32/sizeof(mfloat);
  const uint width = 32+2*pad;
  const uint tye= ti/width;
  const uint txe= ti-tye*width;
  const uint tye2=tye+4;

  int  ixe = blockIdx.x*32 + txe  - pad;
  int  iye = blockIdx.y*6  + tye  - 1;
  int  iye2= blockIdx.y*6  + tye2 - 1;

  // periodicity
  if(ixe<0)       ixe  += dimx;
  if(ixe>dimx-1)  ixe  -= dimx;
  if(iye<0)       iye  += dimy;
  if(iye>dimy-1)  iye  -= dimy;
  if(iye2<0)      iye2 += dimy;
  if(iye2>dimy-1) iye2 -= dimy;

  uint i1, i2;
  
  i1 = ixe+iye*pitch ;
  i2 = ixe+iye2*pitch ;

  mfloat t1 = 0;
  mfloat t2 = 0;
  mfloat t3 = 0;
  mfloat *kernel = d_kernel_3c;
  mfloat C0, C1, C2, C3;
  C0 = kernel[9+4];
  C1 = kernel[4];
  C2 = kernel[1];
  C3 = kernel[0];

  cg::thread_block block = cg::this_thread_block();
  extern __shared__ mfloat shm[];

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  block.sync();  
  push_regs_exp(shm+pad+width, width);  
  block.sync();

  i1 += pitch*pitchy;
  i2 += pitch*pitchy;

  shm[txe +tye *width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  t1 = stencil_3x3_reg(C1, C2, C3);

  block.sync();  
  push_regs_exp(shm+pad+width, width);  
  block.sync();

  i1 += pitch*pitchy;
  i2 += pitch*pitchy;

  shm[txe +tye*width] = in[i1];
  shm[txe+tye2*width] = in[i2];

  t2 = stencil_3x3_reg(C1, C2, C3);
  t1+= stencil_3x3_reg(C0, C1, C2);

  for(uint kk=kstart; kk<kend-1; kk++){

    block.sync();  
    push_regs_exp(shm+pad+width, width);  
    block.sync();

    i1 += pitch*pitchy;
    i2 += pitch*pitchy;

    shm[txe +tye *width] = in[i1];
    shm[txe+tye2*width] = in[i2];

    t3 = stencil_3x3_reg(C1, C2, C3);

    out[ix + iy*pitch + kk*pitch*pitchy] = t1 + t3;
    t1 = t2 + stencil_3x3_reg(C0, C1, C2);
    t2 = t3;
  }

  block.sync();  
  push_regs_exp(shm+pad+width, width);  
  block.sync();

  out[ix + iy*pitch + (kend-1)*pitch*pitchy] = t1 + stencil_3x3_reg(C1, C2, C3);
}
