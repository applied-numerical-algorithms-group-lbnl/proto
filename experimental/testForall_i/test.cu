#include <iostream>
#include <Proto.H>

#include <cstdio>
#include <cstring>
#include <cassert>
#include <cmath>

#include <vector>
#include <memory>

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>

#include "Proto.H"
#include "Proto_WriteBoxData.H"
#include "Proto_Timer.H"

using namespace std;
using namespace Proto;

typedef Var<double,1> State;
typedef Var<double,1> V;


PROTO_KERNEL_START
unsigned int InitF(State& a_U, double a_val)
{
    a_U(0) = a_val;
    return 0;
}
PROTO_KERNEL_END(InitF, Init)

PROTO_KERNEL_START
unsigned int Init_pV2F(Point p, State& a_U, double a_val)
{
    a_U(0) = p[0]+p[1]*10;
    return 0;
}
PROTO_KERNEL_END(Init_pV2F, Init_pV2)


PROTO_KERNEL_START
unsigned int Init_pF(Point p, State& a_U, double a_val)
{
    a_U(0) = a_val;
    return 0;
}
PROTO_KERNEL_END(Init_pF, Init_p)

void WriteData( BoxData<double, 1>&a_state, int it)
{
    char basename[1024];
    sprintf(basename,"euler.%06d",it);

    const char* varnames[1];
    varnames[0] = "martin";
    double origin[DIM];
    for (int ii = 0; ii < DIM; ii++)
    {
        origin[ii] = 0.0;
    }
    WriteBoxData(basename,a_state,varnames,origin,1);
};

void print(double *ptr, unsigned int size1D)
{
  //edge = 1
  for(int i = 0; i<size1D ; i++)
  {
	for(int j = 0 ; j<size1D ; j++)
		std::cout << ptr[i+j*size1D] << " ";
        std::cout << std::endl;
  }				
  std::cout << std::endl;
}

bool checkAnswer(double *ptr, unsigned int size1D)
{
  //edge = 1
  for(int i = 0; i<size1D ; i++)
	for(int j = 0 ; j<size1D ; j++)
		if( (i==0 || i == size1D-1) && (j==0 || j==size1D-1) )
			if(ptr[i+j*size1D]!=1)
			{
				std::cout << " error [" << i << "," << j << "] =" << ptr[i+j*size1D] << " != 1 " <<std::endl;
				return false;
			}
  //inside = 2
  for(int i = 1; i<size1D-1 ; i++)
	for(int j = 1 ; j<size1D-1 ; j++)
		if(ptr[i+j*size1D]!=2)
		{
			std::cout << " error [" << i << "," << j << "] =" << ptr[i+j*size1D] << " != 2 " <<std::endl;
			return false;
		}
  return true;
}


bool checkAnswer_p(double *ptr, unsigned int size1D)
{
  //edge = 1
  for(int i = 0; i<size1D ; i++)
	for(int j = 0 ; j<size1D ; j++)
		if( (i==0 || i == size1D-1) && (j==0 || j==size1D-1) )
			if(ptr[i+j*size1D]!=1)
			{
				std::cout << " error [" << i << "," << j << "] =" << ptr[i+j*size1D] << " != 1 " <<std::endl;
				return false;
			}
  //inside = 2
  for(int i = 1; i<size1D-1 ; i++)
	for(int j = 1 ; j<size1D-1 ; j++)
		if(ptr[i+j*size1D]!=i+j*10)
		{
			std::cout << " error [" << i << "," << j << "] =" << ptr[i+j*size1D] << " != 2 " <<std::endl;
			return false;
		}
  return true;
}

int main()
{
  hipSetDevice(1);
  std::cout << " This code works only on GPU " << std::endl;
  std::cout << " Dim = 2" << std::endl;
  unsigned int size1D = 16;
  unsigned int size2D= size1D*size1D;

  Box b = Box::Cube(size1D);
  Box bminus= b.grow(-1);   

  BoxData<double,1> myBoxDatain(b);
  double a = 1;

  std::cout << " fill from " << b.low() << " to " << b.high() << " with 1 "<<std::endl;
  forallInPlace(Init, b, myBoxDatain, a);
  std::cout << " fill from " << bminus.low() << " to " << bminus.high() << " with 2 "<<std::endl;
  a=2;
  forallInPlace(Init, bminus, myBoxDatain, a);


  double *h_ptr = new double[size2D];
  double *d_ptr = myBoxDatain.dataPtr();
  unsigned int sizeBox = myBoxDatain.box().size();
  assert(size2D == sizeBox);
  unsigned int nBytes = sizeBox * sizeof(double);

  hipMemcpy(h_ptr, d_ptr, nBytes, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  bool check = checkAnswer(h_ptr, size1D);

  if(check)
	std::cout << " The result is correct (forall)" << std::endl;
  else 
	std::cout << " The result is wrong (forall)" << std::endl;

  if(check == false)
	print(h_ptr,size1D);

  BoxData<double,1> myBoxDataForAll_p(b);
  double val=1;
  forallInPlace_p(Init_p, b, myBoxDataForAll_p, val);
  val = 2;
  forallInPlace_p(Init_pV2, bminus, myBoxDataForAll_p, val);

  d_ptr = myBoxDataForAll_p.dataPtr();
  hipMemcpy(h_ptr, d_ptr, nBytes, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  check = checkAnswer_p(h_ptr, size1D);

  if(check)
	std::cout << " The result is correct (forall_p) " << std::endl;
  else 
	std::cout << " The result is wrong (forall_p)" << std::endl;

  if(check == false)
	print(h_ptr,size1D);

#ifdef ZASA
  WriteBoxData(myBoxDatain, 1);
#endif
  return 0;
}


