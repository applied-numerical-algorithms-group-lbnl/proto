#include <iostream>
#include <Proto.H>

#include <cstdio>
#include <cstring>
#include <cassert>
#include <cmath>

#include <vector>
#include <memory>

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>

#include "Proto.H"
#include "Proto_WriteBoxData.H"
#include "Proto_Timer.H"

using namespace std;
using namespace Proto;

typedef Var<double,1> State;
typedef Var<double,1> V;


PROTO_KERNEL_START
unsigned int InitoneF(State& a_U)
{
    a_U(0) = 1;
    return 0;
}
PROTO_KERNEL_END(InitoneF, Initone)


PROTO_KERNEL_START
unsigned int InittwoF(State& a_U)
{
    a_U(0) = 2;
    return 0;
}
PROTO_KERNEL_END(InittwoF, Inittwo)

void WriteData( BoxData<double, 1>&a_state, int it)
{
    char basename[1024];
    sprintf(basename,"euler.%06d",it);

    const char* varnames[1];
    varnames[0] = "martin";
    double origin[DIM];
    for (int ii = 0; ii < DIM; ii++)
    {
        origin[ii] = 0.0;
    }
    WriteBoxData(basename,a_state,varnames,origin,1);
};

int main()
{
  hipSetDevice(1);
  std::cout << " This code works only on GPU " << std::endl;
  std::cout << " Dim = 2" << std::endl;
  unsigned int size1D = 16;
  unsigned int size2D= size1D*size1D;

  Box b = Box::Cube(size1D);
  Box bminus= b.grow(-1);   
  Box bminus2= b.grow(-2);   

  BoxData<double,1> myBoxDatain(b);

 // forallInPlace(Initone, b , myBoxDatain);
 // forallInPlace(Inittwo, bminus , myBoxDatain);

  hipDeviceSynchronize();
  WriteBoxData(myBoxDatain, 1);


  return 0;
}


