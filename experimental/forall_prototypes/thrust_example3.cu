#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <iostream>
template<typename T>
struct absolute_value 
{
  __host__ __device__ T operator()(const T &x) const
  {
    return x < T(0) ? -x : x;
  }
};


int main(void)
{
  int* d_data;
  size_t npts =6;
  hipError_t err;
  hipMalloc(&d_data, npts*sizeof(int));
  {
  thrust::device_ptr<int> devptr(d_data);

    
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    fprintf(stderr, "hipGetLastError() failed at %s:%i : %s\n",
            __FILE__, __LINE__, hipGetErrorString(err));
  }
  
  int value = -42;
  thrust::fill(thrust::device, devptr, devptr + npts, value);
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    fprintf(stderr, "hipGetLastError() failed at %s:%i : %s\n",
            __FILE__, __LINE__, hipGetErrorString(err));
  }
  }
/**/

  thrust::device_ptr<int> devptr(d_data);
  int result = thrust::transform_reduce(devptr, devptr + npts,
                                        absolute_value<int>(),
                                        0,
                                        thrust::maximum<int>());
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    fprintf(stderr, "hipGetLastError() failed at %s:%i : %s\n",
            __FILE__, __LINE__, hipGetErrorString(err));
  }
  std::cout << "max value = " << result << std::endl;

/**/
  hipFree(d_data);
  err = hipGetLastError();
  if (err != hipSuccess)
  {
    fprintf(stderr, "hipGetLastError() failed at %s:%i : %s\n",
            __FILE__, __LINE__, hipGetErrorString(err));
  }
  
}

