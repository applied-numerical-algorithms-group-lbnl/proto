#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <cmath>

#include <vector>
#include <memory>

#include <iostream>
#include <fstream>
#include <sstream>
#include "../../include/Proto.H"


using namespace std;
using namespace Proto;

typedef Var<double,   1> Scalar;


PROTO_KERNEL_START void initParabolaT(Point& p, Var<double>& data)
{
  data(0) = 0;
  for(int idir = 0; idir < DIM; idir ++)
  {
    data(0) += p[idir]*p[idir];
  }
}
PROTO_KERNEL_END(initParabolaT, initParabola);
/****************/
PROTO_KERNEL_START void setRHSF(Point&   a_p,  Var<double>& a_rhs)
{
  a_rhs(0) = 1.;
}
PROTO_KERNEL_END(setRHSF, setRHS) 

/****************/
void
multigridSolve()
{
  int nx = 16;
  Point lo = Point::Zeros();
  Point hi = Point::Ones(nx - 1);
  Bx domain(lo, hi);
  BoxData<double> rhs = forall_p<double>(setRHS, domain);
//  BoxData<double> rhs(domain);
//  forallInPlace_p(initParabola, domain, rhs);


  cout << "after setting rhs max  =  "<< rhs.max() << ", min = "<< rhs.min() << endl;

#ifdef PROTO_CUDA
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    fprintf(stderr, "hipGetLastError() failed at %s:%i : %s\n",
            __FILE__, __LINE__, hipGetErrorString(err));
  }
#endif

/**/
}
int main(int argc, char* argv[])
{

  multigridSolve();

}  
