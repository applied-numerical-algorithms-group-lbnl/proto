

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <functional>


__device__
void pointInitMultiple(int idx,  int* a, int* b, int* c)
{
  a[idx]=0;
  b[idx]=idx;
  c[idx]=idx;
}


template <typename Func, typename... Rest >
__global__
void forall(int begin, int N, Func loop_body, Rest*... a)
{
  typename std::remove_reference<Func>::type body{loop_body};
  for(int idx = begin; idx < N; idx++)
    {
      body(idx, a ...);
    }
}

__global__
void pointInit2(int* a, int* b, int* c)
{
//  int index = threadIdx.x;
//  //number of threads in the block
//  int stride = blockDim.x;
  int idx = threadIdx.x;
  a[idx] = 0; 
  b[idx] = idx;
  c[idx] = idx;
}

template <typename Func, typename... Rest >
void forallbvs(int begin, int N, Func loop_body, Rest*... a)
{
  loop_body<<<1, N>>>(a...);
}

int main(int argc, char** argv) 
{
  int n = 2048;

  int* aye, *bee, *cee;
  hipMalloc(&aye, n*sizeof(int));
  hipMalloc(&bee, n*sizeof(int));
  hipMalloc(&cee, n*sizeof(int));


  printf("made it to first forall\n");

  //  forall<<< 1, 1>>> (0, n, &pointInitMultiple, aye, bee, cee);
   forallbvs(0, n, &pointInit2, aye, bee, cee);

  printf("going into cudaSynchronize \n");

  //wait for gpu to finish before going back to cpu stuff
  hipDeviceSynchronize();

  printf("out of cudaSynchronize \n");

int* a, *b, *c;
a = new int[n];
b = new int[n];
c = new int[n];
size_t bytes = n*sizeof(int);
  hipMemcpy(a, aye, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(b, bee, bytes, hipMemcpyDeviceToHost);
  hipMemcpy(c, cee, bytes, hipMemcpyDeviceToHost);
  int a0 = a[0];
  int b0 = b[0];
  int c0 = c[0];


//  int a0 = aye[0];
//  int b0 = bee[0];
//  int c0 = cee[0];

  printf(" a0= %i, b0= %i, c0 = %i\n", a0, b0, c0);


  for(int i=0; i<n; ++i) 
    {
      //printf("i = %i, a= %i, b= %i, c = %i\n",i,  aye[i], bee[i], cee[i]);
      printf("i = %i, a= %i, b= %i, c = %i\n",i,  a[i], b[i], c[i]);
    }

  hipFree(aye);
  hipFree(bee);
  hipFree(cee);

  return 0;
}
