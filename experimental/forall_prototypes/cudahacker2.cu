#include "hip/hip_runtime.h"
#include <stdio.h>

template <typename T>
__global__ void ckernel1(T *data){

  int my_val = (int)(*data+1);
  printf("hello: %d \n", my_val);
}
template <typename TFunc, typename... TArgs>
__global__ void Test(TFunc func, int count, TArgs... args)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 350)
  (*func)<< <1, 1 >> >(args...);
#else
  printf("What are you doing here!?\n");
#endif
}

template <typename... TArgs>
__host__ void Iterate(void(*kernel)(TArgs...), const int sysInfo, int count, TArgs... args)
{
  if(sysInfo >= 350)
  {
    printf("Iterate on GPU\n");
    Test << <1, 1 >> >(kernel, count, args...);
  }
  else
  {
    printf("Iterate on CPU\n");
    Test << <1, 1 >> >(kernel, count, args...);
  }
}

template <typename T>
__global__ void extractor(void (**kernel)(T *)){

  *kernel = ckernel1<T>;
}

template <typename T>
void run_test(T init)
{

  void (*h_ckernel1)(T *);
  void (**d_ckernel1)(T *);
  T *d_data;
  hipMalloc(&d_ckernel1, sizeof(void *));
  hipMalloc(&d_data, sizeof(T));
  hipMemcpy(d_data, &init, sizeof(T), hipMemcpyHostToDevice);
  extractor<<<1,1>>>(d_ckernel1);
  hipMemcpy((void *)&h_ckernel1, (void *)d_ckernel1, sizeof(void *), hipMemcpyDeviceToHost);
  Iterate(h_ckernel1, 350, 1, d_data);
  hipDeviceSynchronize();
  hipFree(d_ckernel1);
  hipFree(d_data);
  return;
}

int main()
{

  run_test(1);
  run_test(2.0f);

  return 0;
}
