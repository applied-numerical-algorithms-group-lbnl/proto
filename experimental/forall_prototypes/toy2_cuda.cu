
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

__device__ int f() { return 42 ; }

__device__ decltype(&f) f_ptr = f ;

template<typename Func>
__global__ void kernel ( Func func )
{
    int k = func () ;
    printf ("%d\n", k) ;
}

// generic mapper to translate all function signatures
template<typename Func>
inline Func mapper(const Func& device_f)
{
  Func rtn(device_f);
  if (hipSuccess != hipMemcpyFromSymbol (&rtn, HIP_SYMBOL(device_f), sizeof (Func)))
    printf ("FAILED to get SYMBOL\n");
  return rtn;
}
  
int main ()
{
 
  kernel <<<1,1>>> (mapper(f_ptr)) ;
    if (hipDeviceSynchronize() != hipSuccess)
        printf ("FAILED\n");
    else
        printf ("SUCCEEDED\n");
}
