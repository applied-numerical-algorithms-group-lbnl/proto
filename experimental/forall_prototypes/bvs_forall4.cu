#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cstdio>
#include <functional>
#include <iostream>
#include <Proto_gpu.H>

/* forall header material ============================ */
template<typename Func, typename... Rest>
__global__
void indexer(int begin, int end, Func body, Rest... a)
{
  int idx = threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<end)
  {
     body(idx, a...);
  }
}
// generic mapper to translate all function signatures
template<typename Func>
inline Func mapper(const Func& device_f)
{
  Func rtn(device_f); // trick needed for lambdas, since lambdas lack null constructors
  if (protoSuccess != protoMemcpyFromSymbol (&rtn, (const void*) device_f, sizeof (Func), 0, protoMemcpyDeviceToHost))
    printf ("FAILED to get SYMBOL\n");
  return rtn;
}

template<typename Func, typename... Rest>
inline
void
forall(int begin, int end, const Func& loop_body, Rest&&... a)
{
  constexpr int stride=8;
  const int blocks = (end-begin)/stride+1;
  protoLaunchKernel(indexer<Func,Rest...>, stride, blocks, begin, end, mapper(loop_body), std::forward<Rest>(a)...);
}

// User pointwise function
__device__ void initMultiF(int idx, int* a, int* b, int* c)
{
  a[idx]=0; b[idx]=idx; c[idx]=idx;
}
__device__ decltype(&initMultiF) initMulti = initMultiF;



// user application code

int main(int argc, char** argv) 
{
  constexpr int n = 16;

  int* dbuffer;  protoMalloc(&dbuffer, 3*n*sizeof(int));
  int* aye=dbuffer, *bee=dbuffer+n, *cee=dbuffer+2*n;
  int hbuffer[3*n];
  int* a=hbuffer, *b=hbuffer+n, *c=hbuffer+2*n;

  forall(0, n, initMulti, aye, bee, cee);

//  FORALL(0, n, initMultiF, aye, bee, cee);
  
  protoMemcpy(hbuffer, dbuffer, 3*n*sizeof(int), protoMemcpyDeviceToHost);

  bool pass=true;
  for(int i=0; i<n; ++i) 
    {
      if(a[i]!= 0 || b[i]!=i || c[i]!=i) pass=false;
      printf("i = %i, a= %i, b= %i, c = %i\n",i,  a[i], b[i], c[i]);
    }
  if(pass) printf("PASS init\n");
  else     printf("FAIL init\n");
 
 
  protoFree(dbuffer);
  

  return 0;
}
