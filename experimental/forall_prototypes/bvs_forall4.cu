

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <functional>
#include <iostream>

/* forall header material ============================ */
template<typename Func, typename... Rest>
__global__
void indexer(int begin, int end, Func body, Rest... a)
{
  int idx = threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<end)
  {
     body(idx, a...);
  }
}
// generic mapper to translate all function signatures
template<typename Func>
inline Func mapper(const Func& device_f)
{
  Func rtn(device_f); // trick needed for lambdas, since lambdas lack null constructors
  if (hipSuccess != hipMemcpyFromSymbol (&rtn, HIP_SYMBOL(device_f), sizeof (Func)))
    printf ("FAILED to get SYMBOL\n");
  return rtn;
}

template<typename Func, typename... Rest>
inline
void
forall(int begin, int end, const Func& loop_body, Rest&&... a)
{
  constexpr int stride=8;
  const int blocks = (end-begin)/stride+1;
  indexer<<<stride, blocks>>>(begin, end, mapper(loop_body), std::forward<Rest>(a)...);
}

// User pointwise function
__device__ void initMultiF(int idx, int* a, int* b, int* c)
{
  a[idx]=0; b[idx]=idx; c[idx]=idx;
}
__device__ decltype(&initMultiF) initMulti = initMultiF;



// user application code

int main(int argc, char** argv) 
{
  constexpr int n = 16;

  int* dbuffer;  hipMalloc(&dbuffer, 3*n*sizeof(int));
  int* aye=dbuffer, *bee=dbuffer+n, *cee=dbuffer+2*n;
  int hbuffer[3*n];
  int* a=hbuffer, *b=hbuffer+n, *c=hbuffer+2*n;

  forall(0, n, initMulti, aye, bee, cee);

//  FORALL(0, n, initMultiF, aye, bee, cee);
  
  hipMemcpy(hbuffer, dbuffer, 3*n*sizeof(int), hipMemcpyDeviceToHost);

  bool pass=true;
  for(int i=0; i<n; ++i) 
    {
      if(a[i]!= 0 || b[i]!=i || c[i]!=i) pass=false;
      printf("i = %i, a= %i, b= %i, c = %i\n",i,  a[i], b[i], c[i]);
    }
  if(pass) printf("PASS init\n");
  else     printf("FAIL init\n");
 
 
  hipFree(dbuffer);
  

  return 0;
}
