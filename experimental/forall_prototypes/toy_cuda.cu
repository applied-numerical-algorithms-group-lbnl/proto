#include "hip/hip_runtime.h"
#include <cstdio>
#include <Proto_gpu.H>

//typedef int (*funcptr) ();

__device__ int f() { return 42 ; }

typedef decltype(&f) funcptr;

__device__ funcptr f_ptr = f ;

__global__ void kernel ( funcptr func )
{
    int k = func () ;
    printf ("%d\n", k) ;
}


int main ()
{
    funcptr h_funcptr ;

    if (protoSuccess != protoMemcpyFromSymbol (&h_funcptr, (const void *)f_ptr, sizeof (funcptr), 0, protoMemcpyDeviceToHost))
        printf ("FAILED to get SYMBOL\n");

    protoLaunchKernel(kernel, 1, 1, (h_funcptr)) ;
    if (protoDeviceSynchronize() != protoSuccess)
        printf ("FAILED\n");
    else
        printf ("SUCCEEDED\n");
}
