#include "hip/hip_runtime.h"
const int N = 1 << 25;

#include <iostream>
#include <pthread.h>
#include <omp.h>
#include <thread>

#include <chrono>
#include <thread>
#include "Proto_gpu.H"

__global__ void kernel(double *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < 80; i ++) {
        x[tid] = sqrt(pow(3.14159,i));
    }
}

__global__ void kernel2(double *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j = tid; j < n; j+=blockDim.x * gridDim.x)
    for (int i = 0; i < 80; i ++) {
        x[j] = sqrt(pow(3.14159,i));
    }
}



void launch_kernel(protoStream_t *streams, double **d_data, int i, int N)
{
	protoLaunchKernelMemAsync(kernel, N/1024, 1024, 0, streams[i], d_data[i], N);
	return;
}


void launch_copyhtod(protoStream_t *streams, double **h_data,  double **d_data, int i, int N)
{
	protoMemcpyAsync(d_data[i], h_data[i], N, protoMemcpyHostToDevice,streams[i]);
	return;
}

void launch_copydtoh(protoStream_t *streams, double **h_data,  double **d_data, int i, int N)
{
	protoMemcpyAsync(h_data[i], d_data[i], N, protoMemcpyDeviceToHost,streams[i]);
	return;
}

int main()
{
    const int num_streams = 4;

    protoStream_t streams[num_streams];

    std::thread threads[num_streams];



    for (int i = 0; i < num_streams; i++) {
        protoStreamCreate(&streams[i]);
    }

    double** h_data=   new double*[num_streams];
    double* d_data[num_streams];

   for(int i=0 ; i< num_streams; i++) {

     h_data[i] = new double[N];
     for(int j=0; j<N; j++)
	h_data[i][j]=1.;

     protoMalloc(d_data[i], N * sizeof(double));
   }

    
    int nbLoop = 3;

std::cout << " Init " << std::endl;
	

	for(int f=0; f<nbLoop; f++)
	{
		#pragma omp parallel for
		for(int i=0 ; i< num_streams; i++)     
		{  
			protoMemcpyAsync(d_data[i], h_data[i], N, protoMemcpyHostToDevice,streams[i]);
		}

		#pragma omp parallel for
        	for(int i=0 ; i< num_streams; i++)  
		{
			protoLaunchKernelMemAsync(kernel, N/64, 64, 0, streams[i], d_data[i], N);
		}

		#pragma omp parallel for
 		for(int i=0 ; i< num_streams; i++)
		{
			protoMemcpyAsync(h_data[i], d_data[i], N, protoMemcpyDeviceToHost,streams[i]); 
		}
		
	}


	protoDeviceSynchronize();
	std::this_thread::sleep_for(std::chrono::milliseconds(200));

	for(int f=0; f<nbLoop; f++)
	{
		#pragma omp parallel for
        	for(int i=0 ; i< num_streams; i++)  
		{      
			protoMemcpyAsync(d_data[i], h_data[i], N, protoMemcpyHostToDevice,streams[i]);

			protoLaunchKernelMemAsync(kernel, N/64, 64, 0, streams[i], d_data[i], N);

			protoMemcpyAsync(h_data[i], d_data[i], N, protoMemcpyDeviceToHost,streams[i]); 
		}

		
	}

    protoDeviceReset();

    return 0;
}
