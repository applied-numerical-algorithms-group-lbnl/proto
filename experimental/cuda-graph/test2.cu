
#include <hip/hip_runtime.h>
#include <iostream>

#define N 500000 
#define NSTEP 1000
#define NKERNEL 20

__global__ 
void shortKernel(float * out_d, float * in_d){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<N) out_d[idx]=1.23*in_d[idx];
}

__global__
void bigKernel(float * out_d, float * in_d){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<N)
  {
    for(int it = 0; it<NKERNEL ; it++)
      out_d[idx]=1.23*in_d[idx];
  }
}

__global__ 
void initKernel(float * out_d, float * in_d){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<N) out_d[idx]=0;
  if(idx<N) in_d[idx] =1;
}

int main()
{
  float * in_d, *out_d;
  hipMalloc(&in_d, N*sizeof(float));
  hipMalloc(&out_d, N*sizeof(float));

  unsigned int threads = 256;
  unsigned int blocks = (N+threads-1)/threads;

  initKernel<<<blocks, threads, 0, 0>>>(out_d, in_d);

  std::cout << " classic " << std::endl;
  hipEvent_t start, stop;

  {
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // start CPU wallclock timer
    for(int istep=0; istep<NSTEP; istep++){
      for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
        shortKernel<<<blocks, threads, 0, 0>>>(out_d, in_d);
        hipStreamSynchronize(0);
      }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << " time: " << milliseconds << " ms" << std::endl;
  }
  //end CPU wallclock time

  std::cout << " graph " << std::endl;
  initKernel<<<blocks, threads, 0, 0>>>(out_d, in_d);
  bool graphCreated=false;
  hipGraph_t graph;
  hipGraphExec_t instance;
  {
    hipStream_t stream;
    hipStreamCreate ( &stream);
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for(int istep=0; istep<NSTEP; istep++){
      if(!graphCreated)
      {
        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
        for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
          shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
        }
        hipStreamEndCapture(stream, &graph);
        hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
        graphCreated=true;
      }
      hipGraphLaunch(instance, stream);
      hipStreamSynchronize(stream);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << " time: " << milliseconds << " ms" << std::endl;
  }

  std::cout << " Fusion " << std::endl;
  initKernel<<<blocks, threads, 0, 0>>>(out_d, in_d);
  {
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // start CPU wallclock timer
    for(int istep=0; istep<NSTEP; istep++){
      bigKernel<<<blocks, threads, 0, 0>>>(out_d, in_d);
      hipStreamSynchronize(0);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << " time: " << milliseconds << " ms" << std::endl;
  }

  return 0;
}
